#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "layout.h"

__global__ void layout(node* nodes, unsigned char* edges, int numNodes, int width, int height, int iterations){
  int me = threadIdx.x;
  
  float fx, fy;
  for(int z=0;z<iterations;z++){
    for(int i =0; i < numNodes; i++){
      if( i == me){
	continue;
      }
      fx = fy = 0;
      //Work out the repulsive coulombs law force
      float dx = nodes[me].x - nodes[i].x;
      float dy = nodes[me].y - nodes[i].y;
      float dist = sqrtf(dx*dx + dy *dy);
      
      float ke = 0.05;
      float q1 = 3, q2 = 3;

      if(dist < 5 || !isfinite(dist)){
	dist = 5;
      }
      float f = ke*q1*q2/ (dist*dist);
      //printf("%d", f);
      if(isfinite(f)){
	fx = dx * f;
	fy = dy * f;
      }
      
      if(edges[i + me * numNodes]){
	//Attractive spring force
	//float naturalDistance = nodes[i].width + nodes[me].height; //TODO different sizes
	float naturalWidth = nodes[i].width;
	float naturalHeight = nodes[i].height;
	float f = 2;
	fx += (-f) * (dx - naturalWidth);
	fy += (-f) * (dy - naturalHeight);      
      }
      //Move
      //F=ma => a = F/m
      float mass = 2000;
      float ax = fx / mass;
      float ay = fy / mass;
      if(ax > width/3){
	ax = width/3;
      }else if(ax < -width/3){
	ax = -width/3;
      }else if(!isfinite(ax)){
	ax = 0;
      }
      
      if(ay > height/3){
	ay = height/3;
      }else if(ay < -height/3){
	ay = -height/3;
      }else if(!isfinite(ay)){
	ay = 0;
      }

      nodes[me].nextX = nodes[me].x + nodes[me].dx;
      nodes[me].nextY = nodes[me].y + nodes[me].dy;
      nodes[me].nextdy =nodes[me].dy + ay;
      nodes[me].nextdx =nodes[me].dx + ax;
      
      //Update
      nodes[me].x = nodes[me].nextX;
      nodes[me].y = nodes[me].nextY;
      nodes[me].dx = nodes[me].nextdx;
      nodes[me].dy = nodes[me].nextdy;
    }
  } 
}


void graph_layout(graph* g, int width, int height, int iterations){
  /*
    need to allocate memory for nodes and edges on the device
  */
  unsigned char* edges_device;
  node* nodes_device;
  hipError_t err;

  err = hipMalloc(&edges_device, sizeof(unsigned char)* g->numNodes* g->numNodes);
  if(err != hipSuccess){
    printf("Memory allocation for edges failed\n");
    return;
  }
  
  err = hipMalloc(&nodes_device, sizeof(node) * g->numNodes);
  if(err != hipSuccess){
    printf("Memory allocation for nodes failed\n");
    return;
  }
  
  /* copy data to device */
  err = hipMemcpy(edges_device, g->edges, sizeof(unsigned char)* g->numNodes* g->numNodes, hipMemcpyHostToDevice);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy edges to device\n");
  }

  err = hipMemcpy(nodes_device, g->nodes, sizeof(node)* g->numNodes, hipMemcpyHostToDevice);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy nodes to device\n");
  }

  
  /*COMPUTE*/
  int nt = g->numNodes;
  layout<<<1,nt>>>(nodes_device, edges_device, g->numNodes,width,height, iterations);
  
  /*After computation you must copy the results back*/
  err = hipMemcpy(g->nodes, nodes_device, sizeof(node)* g->numNodes, hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy nodes to device\n");
  }
  
  
  
  /*
    All finished, free the memory now
  */
  err = hipFree(nodes_device);
  if(err != hipSuccess){
    printf("Freeing nodes failed\n");
  }
  
  err = hipFree(edges_device);
  if(err != hipSuccess){
    printf("Freeing edges failed\n");
  }
  
}

