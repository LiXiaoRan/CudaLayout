#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "layout.h"

__global__ void layout(node* nodes, unsigned char* edges, int numNodes, int width, int height, int interations){
  int me = threadIdx.x;
  
  float fx, fy;
  for(int i =0; i < numNodes; i++){
    fx = fy = 0;
    //Work out the repulsive coulombs law force
    float dx = nodes[i].x - nodes[me].x;
    float dy = nodes[i].y - nodes[me].y;
    float dist = sqrtf(dx*dx + dy *dy);

    float ke = 50000;
    float q1 = 3, q2 = 3;
    float f = ke*q1*q2/ (dist*dist);
     
    fx = dx * f;
    fy = dx * y;
    
    if(edges[i + me * numNodes]){
      //Attractive spring force
      //      float naturalDistance = nodes[i].width + nodes[me].height; //TODO different sizes
      float naturalWidth = nodes[i].width;
      float naturalHeight = nodes[i].height;
      float f = 0.2;
      fx += (-f) * (dx - naturalWidth);
      fy += (-f) * (dy - naturalHeight);      
    }
    //Move
    //F=ma => a = F/m
    float mass = 2;
    float ax = fx / mass;
    float ay = fy / mass;
    
    nodes[me].nextdy =nodes[me].dy + ay;
    nodes[me].nextdx =nodes[me].dx + ax;
    nodes[me].nextX = nodes[me].x + dx;
    nodes[me].nextY = nodes[me].y + dy;
    //Update
  }

}


void graph_layout(graph* g, int width, int height, int iterations){
  /*
    need to allocate memory for nodes and edges on the device
  */
  unsigned char* edges_device;
  node* nodes_device;
  hipError_t err;

  err = hipMalloc(&edges_device, sizeof(unsigned char)* g->numNodes* g->numNodes);
  if(err != hipSuccess){
    printf("Memory allocation for edges failed\n");
    return;
  }
  
  err = hipMalloc(&nodes_device, sizeof(node) * g->numNodes);
  if(err != hipSuccess){
    printf("Memory allocation for nodes failed\n");
    return;
  }
  
  /* copy data to device */
  err = hipMemcpy(edges_device, g->edges, sizeof(unsigned char)* g->numNodes* g->numNodes, hipMemcpyHostToDevice);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy edges to device\n");
  }

  err = hipMemcpy(nodes_device, g->nodes, sizeof(node)* g->numNodes, hipMemcpyHostToDevice);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy nodes to device\n");
  }

  
  /*COMPUTE*/
  int nt = g->numNodes;
  layout<<<1,nt>>>(nodes_device, edges_device, g->numNodes,width,height, iterations);
  
  /*After computation you must copy the results back*/
  err = hipMemcpy(g->nodes, nodes_device, sizeof(node)* g->numNodes, hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    printf("Error return from hipMemcpy nodes to device\n");
  }
  
  
  
  /*
    All finished, free the memory now
  */
  err = hipFree(nodes_device);
  if(err != hipSuccess){
    printf("Freeing nodes failed\n");
  }
  
  err = hipFree(edges_device);
  if(err != hipSuccess){
    printf("Freeing edges failed\n");
  }
  
}

